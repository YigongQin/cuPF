#include "hip/hip_runtime.h"
#include "MPIsetting.h"
#include "devicefunc.cu_inl"
#include <algorithm>

void MPIsetting::MPItransferData(int nTimeStep, std::vector<std::pair<float*, int>> fieldChunks)
{
    int blocksize_2d = 128;  // seems reduce the block size makes it a little faster, but around 128 is okay.
    for (auto & field : fieldChunks)
    {
        int threadsRequired = field.second*std::max_element(mGeometrySize.begin(), mGeometrySize.end());
        int dataAcquired = 0;
        int num_block_2d = (threadsRequired + blocksize_2d -1)/blocksize_2d;
        collectData1D<<< num_block_2d, blocksize_2d >>>(field.first, field.second, dataAcquired, 
                                                        nxLocal, nyLocal, nzLocal, haloWidth);
        dataAcquired += field.second*mGeometrySize[0];
    }

    hipDeviceSynchronize();    

    exchangeBoundaryData(nTimeStep); 

    for (auto & field : fieldChunks)
    {
        int threadsRequired = field.second*std::max_element(mGeometrySize.begin(), mGeometrySize.end());
        int dataAcquired = 0;
        int num_block_2d = (threadsRequired + blocksize_2d -1)/blocksize_2d;
        distributeData1D<<< num_block_2d, blocksize_2d >>>(field.first, field.second, dataAcquired,
                                                           nxLocal, nyLocal, nzLocal, haloWidth);
        dataAcquired += field.second*mGeometrySize[0];
    }

    hipDeviceSynchronize();      
}

__global__ void
collectData1D(float* field, int numFields, int offset, 
              int nxLocal, int nyLocal, int nzLocal, int haloWidth)
{
    int C = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j, k, PF_id, fnx, fny, fnz;
    G2L_3D(C, i, j, k, PF_id, nxLocal, nyLocal, nzLocal);
    if ( (i<haloWidth) && (j<nyLocal) && (k<nzLocal) && (PF_id<numFields))
    {
        fnx = nxLocal + haloWidth*2;
        fny = nyLocal + haloWidth*2;
        fnz = nzLocal + haloWidth*2;
        // int field_indexL = i+hd+(j+hd)*fnx;
        int field_indexL = L2G_4D(i + haloWidth, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);
        int field_indexR = L2G_4D(i + nxLocal, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);

        mMPIBuffer["sendL"].first[C + offset] = field[field_indexL];
        mMPIBuffer["sendR"].first[C + offset] = field[field_indexR];
    }
}

__global__ void
distributeData1D(float* field, int numFields, int offset, 
                 int nxLocal, int nyLocal, int nzLocal, int haloWidth)
{
  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int i, j, k, PF_id, fnx, fny, fnz;
  fnx = nxLocal + haloWidth*2;
  fny = nyLocal + haloWidth*2;
  fnz = nzLocal + haloWidth*2;

  G2L_3D(C, i, j, k, PF_id, nxLocal, nyLocal, nzLocal);

  if ( (i<haloWidth) && (j<nyLocal) && (k<nzLocal) && (PF_id<numFields))
  {

      // int field_indexL = i+hd+(j+hd)*fnx;
      int field_indexL = L2G_4D(i, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);
      int field_indexR = L2G_4D(i + nxLocal + haloWidth, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);

      field[field_indexL] = mMPIBuffer["recvL"].first[C + offset];
      field[field_indexR] = mMPIBuffer["recvR"].first[C + offset];
  }
}



