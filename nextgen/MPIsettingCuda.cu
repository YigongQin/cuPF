#include "hip/hip_runtime.h"
#include "MPIsetting.h"
#include "devicefunc.cu_inl"
#include <algorithm>

__global__ void
collectData1D(MPIsetting* p, float* field, int numFields, int offset, float* sendBufferL, float* sendBufferR)
{
    int C = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j, k, PF_id, fnx, fny, fnz;
    int nxLocal = p->nxLocal;
    int nyLocal = p->nyLocal;
    int nzLocal = p->nzLocal;
    int haloWidth = p->haloWidth;
    G2L_3D(C, i, j, k, PF_id, nxLocal, nyLocal, nzLocal);
    if ( (i<haloWidth) && (j<nyLocal) && (k<nzLocal) && (PF_id<numFields))
    {
        fnx = nxLocal + haloWidth*2;
        fny = nyLocal + haloWidth*2;
        fnz = nzLocal + haloWidth*2;

        int field_indexL = L2G_4D(i + haloWidth, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);
        int field_indexR = L2G_4D(i + nxLocal, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);

        sendBufferL[C + offset] = field[field_indexL];
        sendBufferR[C + offset] = field[field_indexR];
    }
}


__global__ void
distributeData1D(MPIsetting* p, float* field, int numFields, int offset, float* recvBufferL, float* recvBufferR)
{
  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int i, j, k, PF_id, fnx, fny, fnz;
  int nxLocal = p->nxLocal;
  int nyLocal = p->nyLocal;
  int nzLocal = p->nzLocal;
  int haloWidth = p->haloWidth;
  G2L_3D(C, i, j, k, PF_id, nxLocal, nyLocal, nzLocal);

  if ( (i<haloWidth) && (j<nyLocal) && (k<nzLocal) && (PF_id<numFields))
  {
      fnx = nxLocal + haloWidth*2;
      fny = nyLocal + haloWidth*2;
      fnz = nzLocal + haloWidth*2;

      int field_indexL = L2G_4D(i, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);
      int field_indexR = L2G_4D(i + nxLocal + haloWidth, j + haloWidth, k + haloWidth, PF_id, fnx, fny, fnz);

      field[field_indexL] = recvBufferL[C + offset];
      field[field_indexR] = recvBufferR[C + offset];
  }
}



void MPIsetting::MPItransferData(int nTimeStep, std::vector<std::pair<float*, int>> fieldChunks)
{
    int blocksize_2d = 128;  // seems reduce the block size makes it a little faster, but around 128 is okay.
    for (auto & field : fieldChunks)
    {
        int threadsRequired = *std::max_element(mGeometrySize.begin(), mGeometrySize.end())*field.second;
        int dataAcquired = 0;
        int num_block_2d = (threadsRequired + blocksize_2d -1)/blocksize_2d;
        collectData1D<<< num_block_2d, blocksize_2d >>>(this, field.first, field.second, dataAcquired, mMPIBuffer["sendL"].first, mMPIBuffer["sendR"].first);
        dataAcquired += field.second*mGeometrySize[0];
    }

    hipDeviceSynchronize();    

    exchangeBoundaryData(nTimeStep); 

    for (auto & field : fieldChunks)
    {
        int threadsRequired = *std::max_element(mGeometrySize.begin(), mGeometrySize.end())*field.second;
        int dataAcquired = 0;
        int num_block_2d = (threadsRequired + blocksize_2d -1)/blocksize_2d;
        distributeData1D<<< num_block_2d, blocksize_2d >>>(this, field.first, field.second, dataAcquired, mMPIBuffer["recvL"].first, mMPIBuffer["recvR"].first);
        dataAcquired += field.second*mGeometrySize[0];
    }

    hipDeviceSynchronize();      
}
