#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "CycleTimer.h"

using namespace std;

// this is dependent on the time tiling and grid size of one thread block
// we first finish a non-time tiling version

#define HALO 1//halo in global region
#define BLOCK_DIM_X 128
#define REAL_DIM 126 //BLOCK_DIM_X-2*HALO
#define SHARESIZE 384

void printCudaInfo();
extern float toBW(int bytes, float sec);

struct GlobalConstants {
  int nx;
  int ny;
  int Mt;
  int nts; 
  int ictype;
  float G;
  float R;
  float delta;
  float k;
  float c_infm;
  float Dl;
  float d0;
  float W0;
  float lT;
  float lamd; 
  float tau0;
  float c_infty; 
  float R_tilde;
  float Dl_tilde; 
  float lT_tilde; 
  float eps; 
  float alpha0; 
  float dx; 
  float dt; 
  float asp_ratio; 
  float lxd;
  float lx; 
  float lyd; 
  float eta; 
  float U0; 
  // parameters that are not in the input file
  float hi;
  float cosa;
  float sina;
  float sqrt2;
  float a_s;
  float epsilon;
  float a_12;

};

__constant__ GlobalConstants cP;

// Device codes 

// boundary condition
// only use this function to access the boundary points, 
// other functions return at the boundary
// TODO: this function is doing what, we can definetly merge this into kenrel right?
__global__ void
set_BC(float* ps, float* ph, float* U, float* dpsi, int fnx, int fny){

  // find the location of boundary:
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // z=0, lx
  if (index<fnx) {
    int b_in = index+2*fnx;
    int t_out = index+(fny-1)*fnx;
    int t_in = index+(fny-3)*fnx;

    ps[index] = ps[b_in];
    ph[index] = ph[b_in];
    U[index] = U[b_in];
    dpsi[index] = dpsi[b_in];

    ps[t_out] = ps[t_in];
    ph[t_out] = ph[t_in];
    U[t_out] = U[t_in];
    dpsi[t_out] = dpsi[t_in];
  }
  if (index<fny){
    int l_out = index*fnx;
    int l_in = index*fnx + 2;
    int r_out = index*fnx + fnx -1;
    int r_in = index*fnx + fnx -3;
 
    ps[l_out] = ps[l_in];
    ph[l_out] = ph[l_in];
    U[l_out] = U[l_in];
    dpsi[l_out] = dpsi[l_in];
 
    ps[r_out] = ps[r_in];
    ph[r_out] = ph[r_in];
    U[r_out] = U[r_in];
    dpsi[r_out] = dpsi[r_in];
  }
}

// initialization
__global__ void
initialize(float* ps_old, float* ph_old, float* U_old, float* ps_new, float* ph_new, float* U_new
           , float* x, float* y, int fnx, int fny){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  // obtain i and j(2D position)
  int j=C/fnx;
  int i=C-j*fnx;
  // when initialize, you need to consider C/F layout
  // if F layout, the 1D array has peroidicity of nx    
  // all the variables should be functions of x and y
  // size (nx+2)*(ny+2), x:nx, y:ny
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
    float xc = x[i];
    float yc = y[j];
    int cent = fnx/2;
    ps_old[C] = 5.625f - sqrtf( (xc-x[cent])*(xc-x[cent]) + yc*yc )/cP.W0 ;
    //if (C<1000){printf("ps %f\n",ps_old[C]);}
    ps_new[C] = ps_old[C];
    U_old[C] = cP.U0;
    U_new[C] = cP.U0;
    ph_old[C] = tanhf(ps_old[C]/cP.sqrt2);
    ph_new[C] = tanhf(ps_new[C]/cP.sqrt2);
  //  if (C<1000){printf("phi %f\n",ph_old[C]);} 
  }
}

// anisotropy functions
__device__ float
atheta(float ux, float uz){
  
   float ux2 = cP.cosa*ux + cP.sina*uz;
         ux2 = ux2*ux2;
   float uz2 = -cP.sina*ux + cP.cosa*uz;
         uz2 = uz2*uz2;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return cP.a_s*( 1.0f + cP.epsilon*(ux2*ux2 + uz2*uz2) / MAG_sq2);}
   else {return 1.0f;}
}


__device__ float
aptheta(float ux, float uz){

   float uxr = cP.cosa*ux + cP.sina*uz;
   float ux2 = uxr*uxr;
   float uzr = -cP.sina*ux + cP.cosa*uz;
   float uz2 = uzr*uzr;
   float MAG_sq = (ux2 + uz2);
   float MAG_sq2= MAG_sq*MAG_sq;
   if (MAG_sq > cP.eps){
         return -cP.a_12*uxr*uzr*(ux2 - uz2) / MAG_sq2;}
   else {return 0.0f;}
}

// psi & phi equation: two dimensions



// psi equation
__global__ void
rhs_psi(float* ps, float* ph, float* U, float* ps_new, float* ph_new, \
        float* y, float* dpsi, int fnx, int fny, int nt ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx; 
  int i=C-j*fnx;
  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
       // find the indices of the 8 neighbors for center
       //if (C==1000){printf("find");}
       int R=C+1;
       int L=C-1;
       int T=C+fnx;
       int B=C-fnx;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float psipjp=( ps[C] + ps[R] + ps[T] + ps[T+1] ) * 0.25f;
        float psipjm=( ps[C] + ps[R] + ps[B] + ps[B+1] ) * 0.25f;
        float psimjp=( ps[C] + ps[L] + ps[T-1] + ps[T] ) * 0.25f;
        float psimjm=( ps[C] + ps[L] + ps[B-1] + ps[B] ) * 0.25f;

        float phipjp=( ph[C] + ph[R] + ph[T] + ph[T+1] ) * 0.25f;
        float phipjm=( ph[C] + ph[R] + ph[B] + ph[B+1] ) * 0.25f;
        float phimjp=( ph[C] + ph[L] + ph[T-1] + ph[T] ) * 0.25f;
        float phimjm=( ph[C] + ph[L] + ph[B-1] + ph[B] ) * 0.25f;

        // if (C==1001){
        //   printf("detailed check of neighbours 2\n");
        //   printf("R: %f ; L:%f ; T: %f ; B: %f \n", psipjp, psipjm, psimjp, psimjm);
        // }
        
        // ============================
        // right edge flux
        // ============================
        float psx = ps[R]-ps[C];
        float psz = psipjp - psipjm;
        float phx = ph[R]-ph[C];
        float phz = phipjp - phipjm;

        float A  = atheta( phx,phz);
        float Ap = aptheta(phx,phz);
        float JR = A * ( A*psx - Ap*psz );
        
        // ============================
        // left edge flux
        // ============================
        psx = ps[C]-ps[L];
        psz = psimjp - psimjm;
        phx = ph[C]-ph[L];
        phz = phimjp - phimjm; 

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JL = A * ( A*psx - Ap*psz );
        
        // ============================
        // top edge flux
        // ============================
        psx = psipjp - psimjp;
        psz = ps[T]-ps[C];
        phx = phipjp - phimjp;
        phz = ph[T]-ph[C];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JT = A * ( A*psz + Ap*psx );

        // ============================
        // bottom edge flux
        // ============================
        psx = psipjm - psimjm;
        psz = ps[C]-ps[B];
        phx = phipjm - phimjm;
        phz = ph[C]-ph[B];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JB = A * ( A*psz + Ap*psx );

         /*# =============================================================
        #
        # 2. EXTRA TERM: sqrt2 * atheta**2 * phi * |grad psi|^2
        #
        # =============================================================
        # d(phi)/dx  d(psi)/dx d(phi)/dz  d(psi)/dz at nodes (i,j)*/
        float phxn = ( ph[R] - ph[L] ) * 0.5f;
        float phzn = ( ph[T] - ph[B] ) * 0.5f;
        float psxn = ( ps[R] - ps[L] ) * 0.5f;
        float pszn = ( ps[T] - ps[B] ) * 0.5f;

        float A2 = atheta(phxn,phzn);
        A2 = A2*A2;
        float gradps2 = (psxn)*(psxn) + (pszn)*(pszn);
        float extra =  -cP.sqrt2 * A2 * ph[C] * gradps2;

        /*# =============================================================
        #
        # 3. double well (transformed): sqrt2 * phi + nonlinear terms
        #
        # =============================================================*/

        float Up = (y[j]/cP.W0 - cP.R_tilde * (nt*cP.dt) )/cP.lT_tilde;

        float rhs_psi = ((JR-JL) + (JT-JB) + extra) * cP.hi*cP.hi + \
                   cP.sqrt2*ph[C] - cP.lamd*(1.0f-ph[C]*ph[C])*cP.sqrt2*(U[C] + Up);

        /*# =============================================================
        #
        # 4. dpsi/dt term
        #
        # =============================================================*/
        float tp = (1.0f-(1.0f-cP.k)*Up);
        float tau_psi;
        if (tp >= cP.k){tau_psi = tp*A2;}
               else {tau_psi = cP.k*A2;}
        
        dpsi[C] = rhs_psi / tau_psi; 
        
        ps_new[C] = ps[C] +  cP.dt * dpsi[C];
        ph_new[C] = tanhf(ps_new[C]/cP.sqrt2);
        // if (C == 1001){
        //   printf("check data ps: %f and ph: %f and dpsi: %f\n", ps_new[C], ph_new[C], dpsi[C]);
        // }
        }
} 

// U equation
__global__ void
rhs_U(float* U, float* U_new, float* ph, float* dpsi, int fnx, int fny ){

  int C = blockIdx.x * blockDim.x + threadIdx.x;
  int j=C/fnx;
  int i=C-j*fnx;
  // if the points are at boundary, return
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
        // find the indices of the 8 neighbors for center
        int R=C+1;
        int L=C-1;
        int T=C+fnx;
        int B=C-fnx;
        float hi = cP.hi;
        float Dl_tilde = cP.Dl_tilde;
        float k = cP.k;
        float nx,nz;
        float eps = cP.eps;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ps's are defined on cell centers
        float phipjp=( ph[C] + ph[R] + ph[T] + ph[T+1] ) * 0.25f;
        float phipjm=( ph[C] + ph[R] + ph[B] + ph[B+1] ) * 0.25f;
        float phimjp=( ph[C] + ph[L] + ph[T-1] + ph[T] ) * 0.25f;
        float phimjm=( ph[C] + ph[L] + ph[B-1] + ph[B] ) * 0.25f;

        float jat    = 0.5f*(1.0f+(1.0f-k)*U[C])*(1.0f-ph[C]*ph[C])*dpsi[C];
        /*# ============================
        # right edge flux (i+1/2, j)
        # ============================*/
        float phx = ph[R]-ph[C];
        float phz = phipjp - phipjm;
        float phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_ip = 0.5f*(1.0f+(1.0f-k)*U[R])*(1.0f-ph[R]*ph[R])*dpsi[R];	
        float UR = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[R])*(U[R]-U[C]) + 0.5f*(jat + jat_ip)*nx;
    	 
    	 
        /* ============================
        # left edge flux (i-1/2, j)
        # ============================*/
        phx = ph[C]-ph[L];
        phz = phimjp - phimjm;
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}
        
        float jat_im = 0.5f*(1.0f+(1.0f-k)*U[L])*(1.0f-ph[L]*ph[L])*dpsi[L];
        float UL = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[L])*(U[C]-U[L]) + 0.5f*(jat + jat_im)*nx;
    	 
    	 
        /*# ============================
        # top edge flux (i, j+1/2)
        # ============================*/     
        phx = phipjp - phimjp;
        phz = ph[T]-ph[C];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;}    	
  
        float jat_jp = 0.5f*(1.0f+(1.0f-k)*U[T])*(1.0f-ph[T]*ph[T])*dpsi[T];      
        
        float UT = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[T])*(U[T]-U[C]) + 0.5f*(jat + jat_jp)*nz;
    	 
    	 
        /*# ============================
        # bottom edge flux (i, j-1/2)
        # ============================*/  
        phx = phipjm - phimjm;
        phz = ph[C]-ph[B];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;} 

        float jat_jm = 0.5f*(1.0f+(1.0f-k)*U[B])*(1.0f-ph[B]*ph[B])*dpsi[B];              
        float UB = hi*Dl_tilde*0.5f*(2.0f - ph[C] - ph[B])*(U[C]-U[B]) + 0.5f*(jat + jat_jm)*nz;
        
        float rhs_U = ( (UR-UL) + (UT-UB) ) * hi + cP.sqrt2 * jat;
        float tau_U = (1.0f+cP.k) - (1.0f-cP.k)*ph[C];

        U_new[C] = U[C] + cP.dt * ( rhs_U / tau_U );

       }
}

// (phi_new , psi_new, U_old, dpsi) with rotten BC  to (phi_old, psi_old, U_new) with rotten BC
__global__ void
merge_PF(float* ps, float* ph, float* U, float* ps_new, float* ph_new, float* U_new, float* dpsi, float* dpsi_new,\
       float* y, int fnx, int fny, int nt ){
  

  // load old data
  __shared__ float ps_shared[SHARESIZE];
  __shared__ float ph_shared[SHARESIZE];
  __shared__ float U_shared[SHARESIZE];
  __shared__ float dpsi_shared[SHARESIZE];
  // write data into new array and update at last
  __shared__ float ps_shared_new[SHARESIZE];
  __shared__ float ph_shared_new[SHARESIZE];
  __shared__ float U_shared_new[SHARESIZE];
  __shared__ float dpsi_shared_new[SHARESIZE];
  
  // local id in thread block
  int tid = threadIdx.x; //0, BLOCK_DIM_X
  
  // block id in core region
  int block_id = blockIdx.x; // 0~num_block_x*num_block_y 


  int block_addr = block_id * REAL_DIM;


  // location in global addr (i, j)
  // add HALO due to global halo region; then minus the halo region in the thread block; last add the local id
  int C = block_addr + HALO - HALO + fnx+ tid; // the according location of the global memory note here you
  // can reach the data only between 0<j<fny-1 
  int j = C/fnx ; 
  int i = C - fnx*j;

  int place = tid + BLOCK_DIM_X;
  
  // load necessary data
  if ((i < fnx) && (j < fny-1) && (j>0) ){
  ps_shared[place] = ps[C];
  ph_shared[place] = ph[C];
  U_shared[place]  = U[C];
  dpsi_shared[place]  = dpsi[C];
  
  ps_shared[place+ BLOCK_DIM_X] = ps[C+fnx];
  ph_shared[place+ BLOCK_DIM_X] = ph[C+fnx];
  U_shared[place+ BLOCK_DIM_X]  = U[C+fnx];
  dpsi_shared[place+ BLOCK_DIM_X]  = dpsi[C+fnx];
  
  ps_shared[place -BLOCK_DIM_X] = ps[C-fnx];
  ph_shared[place-BLOCK_DIM_X] = ph[C-fnx];
  U_shared[place-BLOCK_DIM_X]  = U[C-fnx];
  dpsi_shared[place-BLOCK_DIM_X]  = dpsi[C-fnx];  
  }
  __syncthreads();
  // updaate BC first
  // bottom line
  /*
  if ((j == 2) && (i < fnx) &&(place>=2*BLOCK_DIM_X) ){
      ps_shared[place - 2*BLOCK_DIM_X] = ps_shared[place];
      ph_shared[place - 2*BLOCK_DIM_X] = ph_shared[place];
      dpsi_shared[place - 2*BLOCK_DIM_X] = dpsi_shared[place];
      U_shared[place - 2*BLOCK_DIM_X] = U_shared[place];

      ps[C-2*fnx] = ps_shared[place];
      ph[C-2*fnx] = ph_shared[place];
      dpsi[C-2*fnx] = dpsi_shared[place];
      U[C-2*fnx] = U_shared[place];
  }
  // up line
  if ((j == fny - 3)&& (i < fnx) &&(place<BLOCK_DIM_X)){
      ps_shared[place+2*BLOCK_DIM_X] = ps_shared[place];
      ph_shared[place+2*BLOCK_DIM_X] = ph_shared[place];
      dpsi_shared[place+2*BLOCK_DIM_X]   = dpsi_shared[place];
      U_shared[place+2*BLOCK_DIM_X] = U_shared[place];
      
      ps[C+2*fnx] = ps_shared[place];
      ph[C+2*fnx] = ph_shared[place];
      dpsi[C+2*fnx] = dpsi_shared[place];
      U[C+2*fnx] = U_shared[place];
  }
   
  __syncthreads();
  
  // left line
  if ((i == 0) && (j < fny)){
      ps_shared[place] = ps_shared[place + 2];
      ph_shared[place] = ph_shared[place + 2];
      dpsi_shared[place]   = dpsi_shared[place + 2];
      U_shared[place] = U_shared[place + 2];

      ps[C] = ps_shared[place + 2];
      ph[C] = ph_shared[place + 2];
      dpsi[C] = dpsi_shared[place + 2];
      U[C] = U_shared[place + 2];
  }
  // right line
  if ((i == fnx - 1) && (j < fny)){
      ps_shared[place] = ps_shared[place - 2];
      ph_shared[place] = ph_shared[place - 2];
      dpsi_shared[place]   = dpsi_shared[place - 2];
      U_shared[place] = U_shared[place - 2];

      ps[C] = ps_shared[place - 2];
      ph[C] = ph_shared[place - 2];
      dpsi[C] = dpsi_shared[place - 2];
      U[C] = U_shared[place - 2];
  }
  
  __syncthreads();
  */
  // update U
  if ( (i>0) && (i<fnx-1) && (j>0) && (j<fny-1) ) {
    // only update the inner res
    if ( (0<tid) && (tid<BLOCK_DIM_X -1) ) {
      // find the indices of the 8 neighbors for center
        int R=place+1;
        int L=place-1;
        int T=place+BLOCK_DIM_X;
        int B=place-BLOCK_DIM_X;
        float hi = cP.hi;
        float Dl_tilde = cP.Dl_tilde;
        float k = cP.k;
        float nx, nz;
        float eps = cP.eps;
        // =============================================================
        // 1. ANISOTROPIC DIFFUSION
        // =============================================================

        // these ph's are defined on cell centers
        // these ps's are defined on cell centers
        float psipjp=( ps_shared[place] + ps_shared[R] + ps_shared[T] + ps_shared[T+1] ) * 0.25f;
        float psipjm=( ps_shared[place] + ps_shared[R] + ps_shared[B] + ps_shared[B+1] ) * 0.25f;
        float psimjp=( ps_shared[place] + ps_shared[L] + ps_shared[T-1] + ps_shared[T] ) * 0.25f;
        float psimjm=( ps_shared[place] + ps_shared[L] + ps_shared[B-1] + ps_shared[B] ) * 0.25f;

        float phipjp=( ph_shared[place] + ph_shared[R] + ph_shared[T] + ph_shared[T+1] ) * 0.25f;
        float phipjm=( ph_shared[place] + ph_shared[R] + ph_shared[B] + ph_shared[B+1] ) * 0.25f;
        float phimjp=( ph_shared[place] + ph_shared[L] + ph_shared[T-1] + ph_shared[T] ) * 0.25f;
        float phimjm=( ph_shared[place] + ph_shared[L] + ph_shared[B-1] + ph_shared[B] ) * 0.25f;

        // if (C==1001){
        //   printf("detailed check of neighbours 3\n");
        //   printf("R: %f ; L:%f ; T: %f ; B: %f \n", phipjp, phipjm, phimjp, phimjm);
        // }
        float jat    = 0.5f*(1.0f+(1.0f-k)*U_shared[place])*(1.0f-ph_shared[place]*ph_shared[place])*dpsi_shared[place];
        /*# ============================
        # right edge flux (i+1/2, j)
        # ============================*/
        float phx = ph_shared[R]-ph_shared[place];
        float phz = phipjp - phipjm;
        float phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}

        float psx = ps_shared[R]-ps_shared[place];
        float psz = psipjp - psipjm;

        float A  = atheta( phx,phz);
        float Ap = aptheta(phx,phz);
        float JR = A * ( A*psx - Ap*psz );

        float jat_ip = 0.5f*(1.0f+(1.0f-k)*U_shared[R])*(1.0f-ph_shared[R]*ph_shared[R])*dpsi_shared[R];	
        float UR = hi*Dl_tilde*0.5f*(2.0f - ph_shared[place] - ph_shared[R])*(U_shared[R]-U_shared[place]) + 0.5f*(jat + jat_ip)*nx;
    	 
    	 
        /* ============================
        # left edge flux (i-1/2, j)
        # ============================*/
        phx = ph_shared[place]-ph_shared[L];
        phz = phimjp - phimjm;
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nx = phx / sqrtf(phn2);}
                   else {nx = 0.0f;}

        psx = ps_shared[place]-ps_shared[L];
        psz = psimjp - psimjm;

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JL = A * ( A*psx - Ap*psz );
        
        float jat_im = 0.5f*(1.0f+(1.0f-k)*U_shared[L])*(1.0f-ph_shared[L]*ph_shared[L])*dpsi_shared[L];
        float UL = hi*Dl_tilde*0.5f*(2.0f - ph_shared[place] - ph_shared[L])*(U_shared[place]-U_shared[L]) + 0.5f*(jat + jat_im)*nx;
    	 
    	 
        /*# ============================
        # top edge flux (i, j+1/2)
        # ============================*/     
        phx = phipjp - phimjp;
        phz = ph_shared[T]-ph_shared[place];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;}    	

        psx = psipjp - psimjp;
        psz = ps_shared[T]-ps_shared[place];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JT = A * ( A*psz + Ap*psx );
  
        float jat_jp = 0.5f*(1.0f+(1.0f-k)*U_shared[T])*(1.0f-ph_shared[T]*ph_shared[T])*dpsi_shared[T];      
        
        float UT = hi*Dl_tilde*0.5f*(2.0f - ph_shared[place] - ph_shared[T])*(U_shared[T]-U_shared[place]) + 0.5f*(jat + jat_jp)*nz;
    	 
    	 
        /*# ============================
        # bottom edge flux (i, j-1/2)
        # ============================*/  
        phx = phipjm - phimjm;
        phz = ph_shared[place]-ph_shared[B];
        phn2 = phx*phx + phz*phz;
        if (phn2 > eps) {nz = phz / sqrtf(phn2);}
                   else {nz = 0.0f;} 

        psx = psipjm - psimjm;
        psz = ps_shared[place]-ps_shared[B];

        A  = atheta( phx,phz);
        Ap = aptheta(phx,phz);
        float JB = A * ( A*psz + Ap*psx );

        float jat_jm = 0.5f*(1.0f+(1.0f-k)*U_shared[B])*(1.0f-ph_shared[B]*ph_shared[B])*dpsi_shared[B];              
        float UB = hi*Dl_tilde*0.5f*(2.0f - ph_shared[place] - ph_shared[B])*(U_shared[place]-U_shared[B]) + 0.5f*(jat + jat_jm)*nz;
        
        float rhs_U = ( (UR-UL) + (UT-UB) ) * hi + cP.sqrt2 * jat;
        float tau_U = (1.0f+cP.k) - (1.0f-cP.k)*ph_shared[place];

        U_shared_new[place] = U_shared[place] + cP.dt * ( rhs_U / tau_U );
       // U_new[C] = U_shared_new[place];
        
  __syncthreads();





         /*# =============================================================
        #
        # 2. EXTRA TERM: sqrt2 * atheta**2 * phi * |grad psi|^2
        #
        # =============================================================
        # d(phi)/dx  d(psi)/dx d(phi)/dz  d(psi)/dz at nodes (i,j)*/
        float phxn = ( ph_shared[R] - ph_shared[L] ) * 0.5f;
        float phzn = ( ph_shared[T] - ph_shared[B] ) * 0.5f;
        float psxn = ( ps_shared[R] - ps_shared[L] ) * 0.5f;
        float pszn = ( ps_shared[T] - ps_shared[B] ) * 0.5f;

        float A2 = atheta(phxn,phzn);
        A2 = A2*A2;
        float gradps2 = (psxn)*(psxn) + (pszn)*(pszn);
        float extra =  -cP.sqrt2 * A2 * ph_shared[place] * gradps2;

        /*# =============================================================
        #
        # 3. double well (transformed): sqrt2 * phi + nonlinear terms
        #
        # =============================================================*/

        float Up = (y[j]/cP.W0 - cP.R_tilde * (nt*cP.dt) )/cP.lT_tilde;

        float rhs_psi = ((JR-JL) + (JT-JB) + extra) * cP.hi*cP.hi + \
                   cP.sqrt2*ph_shared[place] - cP.lamd*(1.0f-ph_shared[place]*ph_shared[place])*cP.sqrt2*(U_shared_new[place] + Up);

        /*# =============================================================
        #
        # 4. dpsi/dt term
        #
        # =============================================================*/
        float tp = (1.0f-(1.0f-cP.k)*Up);
        float tau_psi;
        if (tp >= cP.k){tau_psi = tp*A2;}
               else {tau_psi = cP.k*A2;}
        
        dpsi_shared_new[place] = rhs_psi / tau_psi;
        ps_shared_new[place] = ps_shared[place] +  cP.dt * dpsi_shared_new[place];
        ph_shared_new[place] = tanhf(ps_shared_new[place]/cP.sqrt2);

        ps_new[C] = ps_shared_new[place];
        ph_new[C] = ph_shared_new[place];
        dpsi_new[C] = dpsi_shared_new[place];
        U_new[C] = U_shared_new[place];
         }
       }

}

// Host codes for PF computing
void setup(GlobalConstants params, int fnx, int fny, float* x, float* y, float* phi, float* psi,float* U){
  // we should have already pass all the data structure in by this time
  // move those data onto device
  printCudaInfo();
  float* x_device;// = NULL;
  float* y_device;// = NULL;
  // store two for swap behavior
  float* psi_old;// = NULL;
  float* psi_new;// = NULL;
  float* U_old;// = NULL;
  float* U_new;// = NULL;
  float* phi_old;// = NULL;
  float* phi_new;// = NULL;
  float* dpsi;// = NULL;
  float* dpsi_new;
  // allocate x, y, phi, psi, U related params
  int length = fnx*fny;

  hipMalloc((void **)&x_device, sizeof(float) * fnx);
  hipMalloc((void **)&y_device, sizeof(float) * fny);

  hipMalloc((void **)&phi_old,  sizeof(float) * length);
  hipMalloc((void **)&psi_old,  sizeof(float) * length);
  hipMalloc((void **)&U_old,    sizeof(float) * length);
  hipMalloc((void **)&phi_new,  sizeof(float) * length);
  hipMalloc((void **)&psi_new,  sizeof(float) * length);
  hipMalloc((void **)&U_new,    sizeof(float) * length);
  hipMalloc((void **)&dpsi,    sizeof(float) * length);
  hipMalloc((void **)&dpsi_new,    sizeof(float) * length);  


  // set initial params
  hipMemcpy(x_device, x, sizeof(float) * fnx, hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, sizeof(float) * fny, hipMemcpyHostToDevice);
  hipMemcpy(psi_old, psi, sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(phi_old, phi, sizeof(float) * length, hipMemcpyHostToDevice);
  hipMemcpy(U_old, U, sizeof(float) * length, hipMemcpyHostToDevice);

  // pass all the read-only params into global constant
  hipMemcpyToSymbol(HIP_SYMBOL(cP), &params, sizeof(GlobalConstants));

   int blocksize_1d = 128;
   int blocksize_2d = 128;  // seems reduce the block size makes it a little faster, but around 128 is okay.
   int num_block_2d = (fnx*fny+blocksize_2d-1)/blocksize_2d;
   int num_block_1d = (fnx+fny+blocksize_1d-1)/blocksize_1d;
   printf("nx: %d and ny: %d\n", fnx, fny);
   printf("block size %d, # blocks %d\n", blocksize_2d, num_block_2d); 
   // change the 2d block due to we donn't want to include halo region
   int num_block_sh = (fnx*(fny-2) + REAL_DIM - 1) / REAL_DIM;

   printf("blocks: %d and block_size: %d\n", num_block_sh, REAL_DIM);
   
   
   initialize<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, U_new, x_device, y_device, fnx, fny);
   set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_new, dpsi_new, fnx, fny);
   set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_old, dpsi, fnx, fny);
   rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_old, phi_old, U_old, psi_new, phi_new, y_device, dpsi_new, fnx, fny, 0);
   hipDeviceSynchronize();
   double startTime = CycleTimer::currentSeconds();

   for (int kt=0; kt<params.Mt/2; kt++){
  //  printf("time step %d\n",kt);
    set_BC<<< num_block_1d, blocksize_1d >>>(psi_new, phi_new, U_old, dpsi_new, fnx, fny);
    merge_PF<<< num_block_sh, BLOCK_DIM_X >>>(psi_new, phi_new, U_old, psi_old, phi_old, U_new, dpsi_new, dpsi, y_device, \
                    fnx, fny, 2*kt+1);

  //  rhs_U<<< num_block_2d, blocksize_2d >>>(U_old, U_new, phi_new, dpsi_new, fnx, fny);

    // //hipDeviceSynchronize();
  //  rhs_psi<<< num_block_2d, blocksize_2d >>>(psi_new, phi_new, U_new, psi_old, phi_old, y_device, dpsi, fnx, fny, 2*kt+1 );

    // //hipDeviceSynchronize();
    set_BC<<< num_block_1d, blocksize_1d >>>(psi_old, phi_old, U_new, dpsi, fnx, fny);
    merge_PF<<< num_block_sh, BLOCK_DIM_X >>>(psi_old, phi_old, U_new, psi_new, phi_new, U_old, dpsi, dpsi_new, y_device, \
      fnx, fny, 2*kt+2);
  }

   hipDeviceSynchronize();
   double endTime = CycleTimer::currentSeconds();
   printf("time for %d iterations: %f s\n", params.Mt, endTime-startTime);
   hipMemcpy(psi, psi_old, length * sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(phi, phi_old, length * sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(U, U_old, length * sizeof(float),hipMemcpyDeviceToHost);

  hipFree(x_device); hipFree(y_device);
  hipFree(psi_old); hipFree(psi_new);
  hipFree(phi_old); hipFree(phi_new);
  hipFree(U_old); hipFree(U_new);
  hipFree(dpsi); hipFree(dpsi_new); 


}





void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
